#include <stdio.h>
#include <stdlib.h>

#include "kernel.cu"
#include "support.h"

int main(int argc, char** argv)
{
    Timer timer;
    hipError_t cuda_ret;

    // Check if argument is given
    if (argc != 2)
    {
        printf("Usage: %s <input file>\n", argv[0]);
        exit(0);
    }

    // Open input file from argument
    FILE* fin = fopen(argv[1], "r");
    if (fin == NULL)
    {
        printf("Error opening file %s\n", argv[1]);
        exit(1);
    }

    int n = 0;
    // Read number of points in file
    fscanf(fin, "%d", &n);

    // Declare host points array and read in points from input file
    //Point* h_points = (Point*) malloc( sizeof(Point) * n );

    Point *points, *result;
    hipMallocManaged(&points, sizeof(Point) * n);
    hipMallocManaged(&result, sizeof(Point) * n);

    for (unsigned int i = 0; i < n; i++) { fscanf(fin, "%d\t%d", &points[i].x, &points[i].y); }

    // Close input file
    fclose(fin);

    // Start timer
    startTime(&timer);
    convexHull(points, result, n);
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) printf("Error: %s\n", hipGetErrorString(cuda_ret));
    stopTime(&timer);

    // Print time elapsed and write points to output file
    printf("Time elapsed: %f s\n", elapsedTime(timer));
    FILE* fout = fopen("hull.txt", "w");
    fprintf(fout, "%d\n", n);
    for (unsigned int i = 0; i < n; i++) { fprintf(fout, "%d\t%d\n", result[i].x, result[i].y); }
    fclose(fout);

    // Free memory
    //free(h_points);
    hipFree(points);
    hipFree(result);

    return 0;
}